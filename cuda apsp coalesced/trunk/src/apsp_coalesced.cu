#include "hip/hip_runtime.h"
#include "../include/apsp_coalesced.h"


int main(int argc, char **argv) {

  int n,w;
  //int t;
  float p;
  struct timeval startwtime, endwtime;
  double seq_time;

  if (argc != 4) {
    printf("Usage: %s n p w\n"
           " where\n"
           " 2^n  : size of the sides of the matrix \n"
           " p    : probability that 2 vertices are directly connected\n"
           " w    : maximum value of edge weights\n\n"
           //" 2^t  : number of tiles per thread for the 3rd method\n"
    , argv[0]);

    return (1);
  }

  n = (int) pow(2.0,atoi(argv[1]));
  p = atof(argv[2]);
  w = atoi(argv[3]);
  //t = pow(2,atoi(argv[4]));

  float *W, *W2, *W3, *W4, *Ws;
  float *W1C, *W2C, *W3C, *W4C;
  float *W1L, *W2L, *W3L, *W4L;
  float *WT, *WT2, *WT3, *WT4;

  // Allocate matrix in device memory
  size_t W_size = n*n*sizeof(float);
  float *dev_W;
  dev_W = (float *) malloc(W_size);
  hipMalloc(&dev_W, W_size);

  // Allocate transposed matrix in device memory
  float *dev_WT;
  dev_WT = (float *) malloc(W_size);
  hipMalloc(&dev_WT, W_size);

  // Create Adjacency Matrix
  W = makeAdjacency(n,p,w);

  // Copy it a bunch of times.
  W1C = (float *) malloc(W_size);
  W1L = (float *) malloc(W_size);
  W2 = (float *) malloc(W_size);
  W2C = (float *) malloc(W_size);
  W2L = (float *) malloc(W_size);
  W3 = (float *) malloc(W_size);
  W3C = (float *) malloc(W_size);
  W3L = (float *) malloc(W_size);
  W4 = (float *) malloc(W_size);
  W4C = (float *) malloc(W_size);
  W4L = (float *) malloc(W_size);
  Ws = (float *) malloc(W_size);

  memcpy(W1C,W,W_size);
  memcpy(W1L,W,W_size);
  memcpy(W2,W,W_size);
  memcpy(W2C,W,W_size);
  memcpy(W2L,W,W_size);
  memcpy(W3,W,W_size);
  memcpy(W3C,W,W_size);
  memcpy(W3L,W,W_size);
  memcpy(W4,W,W_size);
  memcpy(W4C,W,W_size);
  memcpy(W4L,W,W_size);
  memcpy(Ws,W,W_size);

  // Create transposed matrix W^T
  WT = (float *) malloc(W_size);
  transpose(W,WT,n);

  // Copy it a bunch of times.
  WT2 = (float *) malloc(W_size);
  WT3 = (float *) malloc(W_size);
  WT4 = (float *) malloc(W_size);
  memcpy(WT2,WT,W_size);
  memcpy(WT3,WT,W_size);
  memcpy(WT4,WT,W_size);

  // Display system information
  int nDevices;
  hipGetDeviceCount(&nDevices);
  printf("\n\n");
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
  }


  // Calculate general Problem Size
  int size = n*n;
  printf("\nSize of problem: %d\n\n",size);

  // Print Adjacency Matrix
  // print_2dmatrix(W,n);
  // print_2dmatrix(WT,n);


  // Method 1: One tile per thread, no shared memory.
  //-------------------------------------------------

  // Calculate Grid and Block dimensions.
  int gridSide = n/16;
  if(n%16)
    gridSide++;
  dim3 dimGrid(gridSide,gridSide);
  dim3 dimBlock(16,16);

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_single_tile_no_shared<<<dimGrid, dimBlock>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nMethod 1: One tile per thread, no shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,gridSide);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W,n);

  //------------------------------------------ Method 1 end.

  // COALESCENT Method 1: One tile per thread, no shared memory.
  //------------------------------------------------------------

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W1C, W_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_WT, WT, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    //printf("%d\n",k );
    CFW_single_tile_no_shared<<<dimGrid, dimBlock>>>(dev_W, dev_WT, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W1C, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nCOALESCENT Method 1: One tile per thread, no shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,gridSide);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W1C,n);

  //------------------------------------------ COALESCENT Method 1 end.


  // SINGLE_ROW_COALESCENCE Method 1: One tile per thread, no shared memory.
  //------------------------------------------------------------------------

  // Calculate Grid and Block dimensions.
  int gridSide_x = n/256 + 1;
  int gridSide_y = n;
  dim3 dimGrid_L(gridSide_x,gridSide_y);

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W1L, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_single_tile_no_shared<<<dimGrid_L, 256>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W1L, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nSINGLE_ROW_COALESCENCE Method 1: One tile per thread, no shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W1L,n);

  //------------------------------------------ SINGLE_ROW_COALESCENCE Method 1 end.


  // Method 2: One tile per thread, using shared memory.
  //-------------------------------------------------

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W2, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_single_tile_shared<<<dimGrid,dimBlock>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W2, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);
  printf("\nMethod 2: One tile per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,gridSide);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W2,n);

  //------------------------------------------ Method 2 end.


  // COALESCENT Method 2: One tile per thread, using shared memory.
  //---------------------------------------------------------------

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W2C, W_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_WT, WT2, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    CFW_single_tile_shared<<<dimGrid,dimBlock>>>(dev_W, dev_WT, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W2C, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);
  printf("\nCOALESCENT Method 2: One tile per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,gridSide);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W2C,n);

  //------------------------------------------ COALESCENT Method 2 end.

  // SINGLE_ROW_COALESCENCE Method 2: One tile per thread, using shared memory.
  //-------------------------------------------------

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W2L, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_single_tile_shared<<<dimGrid_L,256>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W2L, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);
  printf("\nSINGLE_ROW_COALESCENCE Method 2: One tile per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W2L,n);

  //------------------------------------------ SINGLE_ROW_COALESCENCE Method 2 end.



  // Method 3: Two tiles per thread, using shared memory.
  //-------------------------------------------------

  // Calculate Grid dimensions.
  dim3 dimGrid_3(gridSide,max(gridSide/2, 1));

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W3, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_two_tiles_shared<<<dimGrid_3,dimBlock>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W3, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nMethod 3: Two tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,max(gridSide/2, 1));
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W3,n);

  //------------------------------------------ Method 3 end.


  // COALESCENT Method 3: Two tiles per thread, using shared memory.
  //----------------------------------------------------------------

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W3C, W_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_WT, WT3, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_two_tiles_shared<<<dimGrid_3,dimBlock>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W3C, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nCOALESCENT Method 3: Two tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,max(gridSide/2, 1));
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W3C,n);

  //------------------------------------------ COALESCENT Method 3 end.



  // SINGLE_ROW_COALESCENCE Method 3: Two tiles per thread, using shared memory.
  //----------------------------------------------------------------------------

  // Calculate Grid dimensions.
  dim3 dimGrid_3L(gridSide_x,gridSide_y/2);

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W3L, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_two_tiles_shared<<<dimGrid_3L,256>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W3L, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nSINGLE_ROW_COALESCENCE Method 3: Two tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y/2);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W3L,n);

  //------------------------------------------ SINGLE_ROW_COALESCENCE Method 3 end.



  // Method 4: Four tiles per thread, using shared memory.
  //-------------------------------------------------

  // Calculate Grid and Block dimensions.
  dim3 dimGrid_4(gridSide,max(gridSide/4, 1));


  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W4, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_four_tiles_shared<<<dimGrid_4,dimBlock>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W4, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nMethod 4: Four tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,max(gridSide/4, 1));
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W4,n);

  //------------------------------------------ Method 4 end.

  // COALESCENT Method 4: Four tiles per thread, using shared memory.
  //-----------------------------------------------------------------

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W4C, W_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_WT, WT4, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    FW_four_tiles_shared<<<dimGrid_4,dimBlock>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W4C, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nCOALESCENT Method 4: Four tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,max(gridSide/4, 1));
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W4C,n);

  //------------------------------------------ COALESCENT Method 4 end.


  // SINGLE_ROW_COALESCENCE Method 4: Four tiles per thread, using shared memory.
  //-----------------------------------------------------------------------------

  // Calculate Grid dimensions.
  dim3 dimGrid_4L(gridSide_x,gridSide_y/4);

  gettimeofday (&startwtime, NULL);

  // Copy matrix from host memory to device memory
  hipMemcpy(dev_W, W4L, W_size, hipMemcpyHostToDevice);

  // Invoke kernel
  for(int k=0;k<n;k++) {
    LFW_four_tiles_shared<<<dimGrid_4L,256>>>(dev_W, n, k);
    hipDeviceSynchronize();
  }

  // Copy matrix back to host memory.
  hipMemcpy(W4L, dev_W, W_size, hipMemcpyDeviceToHost);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
            + endwtime.tv_sec - startwtime.tv_sec);

  printf("\nSINGLE_ROW_COALESCENCE Method 4: Four tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y/4);
  printf("\n  Clock time = %f\n", seq_time);
  //print_2dmatrix(W4L,n);

  //------------------------------------------ Method 4 end.



  // Serial Implementation
  //-------------------------------------------------

  gettimeofday (&startwtime, NULL);
  Floyd_Warshall(Ws,n);
  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
  		      + endwtime.tv_sec - startwtime.tv_sec);

  printf("\n\nSerial implementation clock time = %f\n", seq_time);

  //------------------------------------------ Serial end.



  // Check if parallel methods work correctly.
  int all_correct = 0;

  fflush( stdout );
  if(!memcmp(Ws, W, n*n*sizeof(float))) {
    printf("\n\nMethod 1 works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W1C, n*n*sizeof(float))) {
    printf("\nMethod 1 COALESCENT works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W1L, n*n*sizeof(float))) {
    printf("\nMethod 1 SINGLE_ROW_COALESCENCE works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W2, n*n*sizeof(float))) {
    printf("\n\nMethod 2 works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W2C, n*n*sizeof(float))) {
    printf("\nMethod 2 COALESCENT works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W2L, n*n*sizeof(float))) {
    printf("\nMethod 2 SINGLE_ROW_COALESCENCE works fine.");
    all_correct++;
  }
   if(!memcmp(Ws, W3, n*n*sizeof(float))) {
    printf("\n\nMethod 3 works fine.");
    all_correct++;
  }
   if(!memcmp(Ws, W3C, n*n*sizeof(float))) {
    printf("\nMethod 3 COALESCENT works fine.");
    all_correct++;
  }
   if(!memcmp(Ws, W3L, n*n*sizeof(float))) {
    printf("\nMethod 3 SINGLE_ROW_COALESCENCE works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W4, n*n*sizeof(float))) {
    printf("\n\nMethod 4 works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W4C, n*n*sizeof(float))) {
    printf("\nMethod 4 COALESCENT works fine.");
    all_correct++;
  }
  if(!memcmp(Ws, W4L, n*n*sizeof(float))) {
    printf("\nMethod 4 SINGLE_ROW_COALESCENCE works fine.");
    all_correct++;
  }

  if(all_correct==12)
    printf("\n\nAll methods work correctly.\n");


  // Free allocated memory.
  free(W);
  free(W1C);
  free(W1L);
  free(W2);
  free(W2C);
  free(W2L);
  free(W3);
  free(W3C);
  free(W3L);
  free(W4);
  free(W4C);
  free(W4L);
  free(Ws);
  free(WT);
  free(WT2);
  free(WT3);
  free(WT4);
  hipFree(dev_W);
  hipFree(dev_WT);

  return 0;
}
