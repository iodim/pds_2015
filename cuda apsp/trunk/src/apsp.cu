#include "hip/hip_runtime.h"
#include "../include/apsp.h"

float inf=1.0/0.0;


float rand_factor();
void print_2dmatrix(float *W, int n);
float* makeAdjacency(int n, float p, int w);
void Floyd_Warshall(float *W, int n);

// Kernel functions
__global__ void FW_single_tile_no_shared(float *W, int n, int k);
__global__ void FW_single_tile_shared(float *W, int n, int k);
__global__ void FW_two_tiles_shared(float *W, int n, int k);
__global__ void FW_four_tiles_shared(float *W, int n, int k);




int main(int argc, char **argv) {

int n,w;
//int t;
float p;
struct timeval startwtime, endwtime;
double seq_time;


if (argc != 4) {
  printf("Usage: %s n p w\n"
         " where\n"
         " 2^n  : size of the sides of the matrix \n"
         " p    : probability that 2 vertices are directly connected\n"
         " w    : maximum value of edge weights\n\n"
         //" 2^t  : number of tiles per thread for the 3rd method\n"
  , argv[0]);

  return (1);
}

n = pow(2,atoi(argv[1]));
p = atof(argv[2]);
w = atoi(argv[3]);
//t = pow(2,atoi(argv[4]));


float *W, *W2, *W3, *W4, *Ws;


// Allocate matrix in device memory
size_t W_size = n*n*sizeof(float);
float *dev_W;
dev_W = (float *) malloc(W_size);
hipMalloc(&dev_W, W_size);

// Create Adjacency Matrix
W = makeAdjacency(n,p,w);

// Copy it a bunch of times.
W2 = (float *) malloc(W_size);
W3 = (float *) malloc(W_size);
W4 = (float *) malloc(W_size);
Ws = (float *) malloc(W_size);
memcpy(W2,W,W_size);
memcpy(W3,W,W_size);
memcpy(W4,W,W_size);
memcpy(Ws,W,W_size);


// Display system information
int nDevices;
hipGetDeviceCount(&nDevices);
printf("\n\n");
for (int i = 0; i < nDevices; i++) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, i);
  printf("Device Number: %d\n", i);
  printf("  Device name: %s\n", prop.name);
  printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
}


// Calculate general Problem Size
int size = n*n;
printf("\nSize of problem: %d\n\n",size);

// Print Adjacency Matrix
//print_2dmatrix(W,n);




// Method 1: One tile per thread, no shared memory.
//-------------------------------------------------

// Calculate Grid and Block dimensions.
int gridSide = n/16;
if(n%16)
  gridSide++;
dim3 dimGrid(gridSide,gridSide);
dim3 dimBlock(16,16);


gettimeofday (&startwtime, NULL);

// Copy matrix from host memory to device memory
hipMemcpy(dev_W, W, W_size, hipMemcpyHostToDevice);

// Invoke kernel
for(int k=0;k<n;k++) {
  FW_single_tile_no_shared<<<dimGrid, dimBlock>>>(dev_W, n, k);
  hipDeviceSynchronize();
}

// Copy matrix back to host memory.
hipMemcpy(W, dev_W, W_size, hipMemcpyDeviceToHost);

gettimeofday (&endwtime, NULL);

seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

printf("\nMethod 1: One tile per thread, no shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide,gridSide);
printf("\n  Clock time = %f\n", seq_time);
// print_2dmatrix(W,n);

//------------------------------------------ Method 1 end.




// Method 2: One tile per thread, using shared memory.
//-------------------------------------------------

// Calculate Grid dimensions.
int gridSide_x = max(n/256, 1);
int gridSide_y = n;

dim3 dimGrid_2(gridSide_x,gridSide_y);

gettimeofday (&startwtime, NULL);

// Copy matrix from host memory to device memory
hipMemcpy(dev_W, W2, W_size, hipMemcpyHostToDevice);

// Invoke kernel
for(int k=0;k<n;k++) {
  FW_single_tile_shared<<<dimGrid_2,256>>>(dev_W, n, k);
  hipDeviceSynchronize();
}

// Copy matrix back to host memory.
hipMemcpy(W2, dev_W, W_size, hipMemcpyDeviceToHost);

gettimeofday (&endwtime, NULL);

seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);
printf("\nMethod 2: One tile per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y);
printf("\n  Clock time = %f\n", seq_time);
// print_2dmatrix(W2,n);

//------------------------------------------ Method 2 end.



// Method 3: Two tiles per thread, using shared memory.
//-------------------------------------------------

// Calculate Grid dimensions.
dim3 dimGrid_3(gridSide_x,gridSide_y/2);

gettimeofday (&startwtime, NULL);

// Copy matrix from host memory to device memory
hipMemcpy(dev_W, W3, W_size, hipMemcpyHostToDevice);

// Invoke kernel
for(int k=0;k<n;k++) {
  FW_two_tiles_shared<<<dimGrid_3,256>>>(dev_W, n, k);
  hipDeviceSynchronize();
}

// Copy matrix back to host memory.
hipMemcpy(W3, dev_W, W_size, hipMemcpyDeviceToHost);

gettimeofday (&endwtime, NULL);

seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

printf("\nMethod 3: Two tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y/2);
printf("\n  Clock time = %f\n", seq_time);
//print_2dmatrix(W3,n);

//------------------------------------------ Method 3 end.


// Method 4: Four tiles per thread, using shared memory.
//-------------------------------------------------

// Calculate Grid dimensions.
dim3 dimGrid_4(gridSide_x,gridSide_y/4);

gettimeofday (&startwtime, NULL);

// Copy matrix from host memory to device memory
hipMemcpy(dev_W, W4, W_size, hipMemcpyHostToDevice);

// Invoke kernel
for(int k=0;k<n;k++) {
  FW_four_tiles_shared<<<dimGrid_4,256>>>(dev_W, n, k);
  hipDeviceSynchronize();
}

// Copy matrix back to host memory.
hipMemcpy(W4, dev_W, W_size, hipMemcpyDeviceToHost);

gettimeofday (&endwtime, NULL);

seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

printf("\nMethod 4: Four tiles per thread, using shared memory:\n  Grid_x: %d\n  Grid_y: %d",gridSide_x,gridSide_y/4);
printf("\n  Clock time = %f\n", seq_time);
//print_2dmatrix(W4,n);

//------------------------------------------ Method 4 end.



// Serial Implementation
//-------------------------------------------------

gettimeofday (&startwtime, NULL);
Floyd_Warshall(Ws,n);
gettimeofday (&endwtime, NULL);

seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
		      + endwtime.tv_sec - startwtime.tv_sec);

printf("\n\nSerial implementation clock time = %f\n", seq_time);

//------------------------------------------ Serial end.



// Check of parallel methods work correctly.
int all_correct = 0;

fflush( stdout );
if(!memcmp(Ws, W, n*n*sizeof(float))) {
  printf("\n\nMethod 1 works fine.");
  all_correct++;
}
if(!memcmp(Ws, W2, n*n*sizeof(float))) {
  printf("\nMethod 2 works fine.");
  all_correct++;
}
 if(!memcmp(Ws, W3, n*n*sizeof(float))) {
  printf("\nMethod 3 works fine.");
  all_correct++;
}
if(!memcmp(Ws, W4, n*n*sizeof(float))) {
  printf("\nMethod 4 works fine.");
  all_correct++;
}

if(all_correct==4)
  printf("\n\nAll methods work correctly.\n");


// Free allocated memory.
free(W);
free(W2);
free(W3);
free(W4);
free(Ws);
hipFree(dev_W);

return 0;
}





// Create Adjacency matrix, based on the matlab code given.
float* makeAdjacency(int n, float p, int w)
{
  int i,j;
  float* A;

  srand(time(NULL));
  A = (float *) malloc(n*n*sizeof(float));

  for(i=0;i<n;i++) {
    for(j=0;j<n;j++)
      if(rand_factor() > p)  A[i*n+j] = inf;
      else  A[i*n+j] = rand_factor()*w;
    A[i*n+i] = 0;
  }

  return A;

}


// Returns random number from 0 to 1.
float rand_factor()
{
    return (float)rand() / (float)RAND_MAX ;
}


// Prints a 2d matrix.
void print_2dmatrix(float *W, int n)
{
  int i,j;
  printf("\n\n");
  for(i=0;i<n;i++)
    for(j=0;j<n;j++) {
      printf("%-9f  ",W[i*n+j]);
      if(j==n-1) printf("\n");
    }

}


// Serial Floyd Warshall implementation.
void Floyd_Warshall(float *W, int n)
{
  int i,j,k;
  for(k=0;k<n;k++)
    for(i=0;i<n;i++)
      for(j=0;j<n;j++)
        if(W[i*n+j] > W[i*n+k] + W[k*n+j])
          W[i*n+j] = W[i*n+k] + W[k*n+j];
}


// Parallel method 1: Single tile per thread, without shared memory.
__global__ void FW_single_tile_no_shared(float *W, int n, int k)
{

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if(col >= n)
    return;


  float row_k = W[n*row+k];
  float k_col = W[k*n+col];
  __syncthreads();  // Does not seem to be necessary, code apparently works fine without it.(****)

  if(W[n*row+col] > row_k + k_col)
    W[n*row+col] = row_k + k_col;
}


// Parallel method 2: Single tile per thread, shared memory.
__global__ void FW_single_tile_shared(float *W, int n, int k)
{

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y;
  if(col >= n)
    return;

  __shared__  float row_k;
  if(threadIdx.x==0) row_k = W[n*row+k];
  float k_col = W[k*n+col];
 __syncthreads();

  if(W[n*row+col] > row_k + k_col)
    W[n*row+col] = row_k + k_col;
}


// Parallel method 3: Two tiles per thread, shared memory.
__global__ void FW_two_tiles_shared(float *W, int n, int k)
{

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y;
  if(col >= n)
    return;

  __shared__ float row_k;
  __shared__ float row_k1;
  if(threadIdx.x==0) {
    row_k = W[n*row+k];
    row_k1 = W[n*row+k+n*gridDim.y];
  }
  float k_col = W[k*n+col];
 __syncthreads();

  float alternative = row_k + k_col;
  float alternative1 = row_k1 + k_col;

  if(W[n*row+col] > alternative)
    W[n*row+col] = alternative;
  if(W[n*row+col+n*gridDim.y] > alternative1)
    W[n*row+col+n*gridDim.y] = alternative1;
}



// Parallel method 4: Four tiles per thread, shared memory.
__global__ void FW_four_tiles_shared(float *W, int n, int k)
{

  int i;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y;
  if(col >= n)
    return;

  __shared__ float row_k[4];
  if(threadIdx.x==0) {
    for(i=0; i<4; i++)
      row_k[i] = W[n*row+k+n*gridDim.y*i];
  }
  float k_col = W[k*n+col];
 __syncthreads();

  for(i=0; i<4; i++)
    if(W[n*row+col+n*gridDim.y*i] > row_k[i] + k_col)
        W[n*row+col+n*gridDim.y*i] = row_k[i] + k_col;
}
